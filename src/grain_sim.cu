#include "hip/hip_runtime.h"
#include <cassert>

#include "gpu_grain_types.h"
#include "grain_types.h"
#include "grain_sim.h"
#include <fmt/format.h>

namespace grain {

__device__ void gpu_update_cell(uint32_t* buf, size_t n, size_t turn, size_t x, size_t y) {
    auto idx = x + y * n;
    auto val = buf[idx];
    if (!is_done(val, turn)) {
        //todo switch-case?
        if (is_type(val, GrainType::Blank)) {

        } else if (is_type(val, GrainType::Sand)) {
            if (y != n - 1) {
                if (is_passable(buf[x + (y + 1) * n])) {
                    val = buf[x + (y + 1) * n] & GrainType::MASK_TYPE;
                    buf[x + (y + 1) * n] = mark_done(GrainType::Sand, turn);
                } else if (x != 0
                           && is_passable(buf[x - 1 + (y + 1) * n])) {
                    val = buf[x - 1 + (y + 1) * n];
                    buf[x - 1 + (y + 1) * n] = mark_done(GrainType::Sand, turn);
                } else if (x != n - 1
                           && is_passable(buf[x + 1 + (y + 1) * n])) {
                    val = buf[x + 1 + (y + 1) * n];
                    buf[x + 1 + (y + 1) * n] = mark_done(GrainType::Sand, turn);
                }
            }
        } else if (is_type(val, GrainType::Water)) {
            if (y != n - 1 && is_type(buf[x + (y + 1) * n], GrainType::Blank)) {
                val = GrainType::Blank;
                buf[x + (y + 1) * n] = mark_done(GrainType::Water, turn);
            } else if (y != n - 1 && x != 0
                       && is_type(buf[x - 1 + (y + 1) * n], GrainType::Blank)) {
                val = GrainType::Blank;
                buf[x - 1 + (y + 1) * n] = mark_done(GrainType::Water, turn);
            } else if (y != n - 1 && x != n - 1
                       && is_type(buf[x + 1 + (y + 1) * n], GrainType::Blank)) {
                val = GrainType::Blank;
                buf[x + 1 + (y + 1) * n] = mark_done(GrainType::Water, turn);
            } else if (x != 0
                       && is_type(buf[x - 1 + y * n], GrainType::Blank)) {
                val = GrainType::Blank;
                buf[x - 1 + y * n] = mark_done(GrainType::Water, turn);
            } else if (x != n - 1
                       && is_type(buf[x + 1 + y * n], GrainType::Blank)) {
                val = GrainType::Blank;
                buf[x + 1 + y * n] = mark_done(GrainType::Water, turn);
            }
        }

        val = mark_done(val, turn);
        buf[idx] = val;
    }
}

__global__ void gpu_slow_step(uint32_t *buf, size_t n, uint32_t turn) {
    for (int x = 0; x < n; x++) {
        for (int y = 0; y < n; y++) {
            gpu_update_cell(buf, n, turn, x, y);
        }
    }
}

__global__ void gpu_step(uint32_t* buf, size_t n, uint32_t turn, size_t bx, size_t by) {
    auto sx = bx * 3 + blockIdx.x * blockDim.x * 6 + threadIdx.x * 6;
    auto sy = by * 3 + blockIdx.y * blockDim.y * 6 + threadIdx.y * 6;
    for(size_t dx=0; dx<3; dx++) {
        for(size_t dy=0; dy<3; dy++) {
            auto x = sx + dx;
            auto y = sy + dy;
            if(x < n && y < n) {
#if 0
                uint32_t col;
                if(bx == 0 && by == 0) {
                    col = GrainType::Debug0;
                } else if(bx == 0 && by == 1) {
                    col = GrainType::Debug1;
                } else if(bx == 1 && by == 0) {
                    col = GrainType::Debug2;
                } else {
                    col = GrainType::Debug3;
                }
                buf[x + y * n] = col;
#endif
                gpu_update_cell(buf, n, turn, x, y);
            }
        }
    }
}

__global__ void gpu_sprinkle(uint32_t *out, size_t n, uint32_t value,
                             size_t start_x, size_t start_y, size_t sz) {
    //todo this is very suspicious maybe some overflows but we are compensating later
    auto x = start_x - sz/2 + blockIdx.x * blockDim.x + threadIdx.x;
    auto y = start_y - sz/2 + blockIdx.y * blockDim.y + threadIdx.y;
    if(x < n && y < n && x < start_x + sz && y < start_y + sz) {
        auto dx = start_x - x;
        auto dy = start_y - y;

        if(dx*dx + dy*dy <= sz * sz / 4) {
            auto idx = x + y * n;
            // todo use hiprand
            out[idx] = value;
        }

    }
}

void GrainSim::step(const GPUImage& in, GPUImage& out) {
    // todo find better way to do double buffer
    out = in; // GPU-copy from in to out

    const size_t T = 16;
    const size_t thirds = (m_N + 3 - 1) / 3;
    dim3 threadsPerBlock(T, T);
    dim3 numBlocks((thirds + 2*T - 1) / (2 * T), (thirds + 2*T - 1) / (2 * T));

    // todo this is incorrect, we maybe wasting a full step on a noop
    auto turn = m_frame_count % 2;

    for(size_t i=0; i<m_speed; i++) {
        turn ^= 1;
        // gpu_slow_step<<<1, 1>>>(out.data(), m_N, turn);
        gpu_step<<<numBlocks, threadsPerBlock>>>(out.data(), m_N, turn, 0, 0);
        gpu_step<<<numBlocks, threadsPerBlock>>>(out.data(), m_N, turn, 0, 1);
        gpu_step<<<numBlocks, threadsPerBlock>>>(out.data(), m_N, turn, 1, 0);
        gpu_step<<<numBlocks, threadsPerBlock>>>(out.data(), m_N, turn, 1, 1);
    }

    cuda_assert(hipPeekAtLastError());
}

void GrainSim::sprinkle(grain::GPUImage &image, uint32_t value,
                        size_t x, size_t y, size_t sz) {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((sz + 16 - 1)/16, (sz + 16 - 1)/16);
    gpu_sprinkle<<<numBlocks, threadsPerBlock>>>(image.data(), image.width(), value,
                                                 x+sz/2, y+sz/2, sz);

    cuda_assert(hipPeekAtLastError());

}
}
