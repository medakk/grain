#include "hip/hip_runtime.h"
#include <cassert>

#include "gpu_image.h"

namespace grain {
__global__ void gpu_fill(uint32_t *buf, size_t n,
                         uint32_t val) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x < n && y < n) {
        auto idx = x + y * n;
        buf[idx] = val;
    }
}
__global__ void gpu_fill_block(uint32_t *buf, size_t n,
                               size_t row, size_t col,
                               size_t n_rows, size_t n_cols,
                               uint32_t val) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= row && x < row + n_rows && y >= col && y < col + n_cols) {
        auto idx = x + y * n;
        buf[idx] = val;
    }
}
__global__ void gpu_count(uint32_t *buf, size_t n, uint32_t val, int* out) {
    auto x = blockIdx.x * blockDim.x + threadIdx.x;
    auto y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x < n && y < n) {
        auto idx = x + y * n;
        if(buf[idx] == val) {
            // todo parallel reduction?
            atomicAdd(out, 1);
        }
    }
}

void GPUImage::fill(uint32_t val) {
    // todo don't require this
    assert(m_N%16 == 0);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(m_N/16, m_N/16);
    gpu_fill<<<numBlocks, threadsPerBlock>>>(m_image, m_N, val);

    cuda_assert(hipPeekAtLastError());
}

void GPUImage::fill(size_t row, size_t col, size_t n_rows, size_t n_cols, uint32_t val) {
    // todo don't require this
    assert(m_N%16 == 0);

    //todo yes this is very inefficient. its just to add colors to debug stuff. maybe should
    // just do it on CPU, its managed memory anyway. But where's the fun in that
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(m_N / 16, m_N / 16);
    gpu_fill_block<<<numBlocks, threadsPerBlock>>>(m_image, m_N,
                                                   row, col, n_rows, n_cols,
                                                   val);

    cuda_assert(hipPeekAtLastError());
}

size_t GPUImage::count(uint32_t val) const {
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(m_N / 16, m_N / 16);

    int* d_out;
    cuda_assert(hipMallocManaged(&d_out, sizeof(int)));
    *d_out = 0;

    gpu_count<<<numBlocks, threadsPerBlock>>>(m_image, m_N, val, d_out);

    cuda_assert(hipPeekAtLastError());
    cuda_assert(hipDeviceSynchronize());

    size_t out = *d_out;
    cuda_assert(hipFree(d_out));

    return out;
}

}
